#include <iostream>
#include <hip/hip_runtime.h>

// Error checking macro
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

void printCudaMemoryInfo() {
  size_t free_mem = 0;
  size_t total_mem = 0;
  CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
  std::cout << "Total memory: " << total_mem / (1024 * 1024) << ", Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
}

void testCudaMallocLimits(bool verbose) {
  size_t size = 1024 * 1024 * 1024; // Start with 1 GB
  float* d_ptr = nullptr;

  while (true) {
    // Print CUDA memory info before allocation
    hipError_t err = hipMalloc((void**)&d_ptr, size);
    if (err == hipSuccess) {
      if (verbose) {
        std::cout << "CUDA memory info after allocation:" << std::endl;
        printCudaMemoryInfo();
      }
      std::cout << "hipMalloc succeeded for size: " << size / (1024 * 1024) << " MB" << std::endl;
      if (verbose) {
        std::cout << "hipFree" << std::endl;
      }
      CHECK_CUDA(hipFree(d_ptr));
      size += 1024 * 1024 * 1024; // Increase by 1 GB
      // Print CUDA memory info after allocation
      if (verbose) {
        std::cout << "CUDA memory info after deallocation:" << std::endl;
        printCudaMemoryInfo();
      }
    } else {
      std::cerr << "hipMalloc failed for size: " << size / (1024 * 1024) << " MB" << std::endl;
      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
      break;
    }
  }
}

int main() {
  std::cout << "CUDA memory info before any allocation:" << std::endl;
  printCudaMemoryInfo();

  // Test CUDA malloc limits
  testCudaMallocLimits(false);

  return 0;
}
